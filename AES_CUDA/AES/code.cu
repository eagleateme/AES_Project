#include "hip/hip_runtime.h"
#include "AES.h"
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>

u_char s_box[] = {0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
  0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
  0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
  0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
  0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
  0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
  0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
  0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
  0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
  0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
  0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
  0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
  0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
  0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
  0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};
u_char Rcon[11] = {0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

void Generate_Nonce_Text(u_char *nonceArr, int blockSize, int numBlocks)
{
	//for each block
	for(int count = 0 ; count < numBlocks ; count++)
	{
		u_char counter_1 = 0x01;

		//for each byte per block
		for(int i = blockSize-1; i >= 0 ; i--)
		{

			int temp = nonceArr[blockSize*count+i]+counter_1;

				if(temp == 256) // > 256
				{
					nonceArr[(blockSize*(count+1))+i] = 0x00;// temp%256
					counter_1 = 0x01;//counter = temp/256;
				}
				else
				{
					nonceArr[(blockSize*(count+1))+i] = temp;// nonceArr[(blockSize*(count))+i]+counter_1;
					counter_1 = 0x00;
				}
		}

	}

	//nonceArr[blockSize*numBlocks]= '\0';
}

//Multiplication in GF(2^8)(source: Wiki)
__device__ u_char gmul(u_char a, u_char b){
	u_char p = 0; /* the product of the multiplication */
	while (a && b) {
	        if (b & 1) /* if b is odd, then add the corresponding a to p (final product = sum of all a's corresponding to odd b's) */
	            p ^= a; /* since we're in GF(2^m), addition is an XOR */

	        if (a & 0x80) /* GF modulo: if a >= 128, then it will overflow when shifted left, so reduce */
	            a = (a << 1) ^ 0x11b; /* XOR with the primitive polynomial x^8 + x^4 + x^3 + x + 1 (0b1_0001_1011) – you can change it but it must be irreducible */
	        else
	            a <<= 1; /* equivalent to a*2 */
	        b >>= 1; /* equivalent to b // 2 */
			}
	return p;}

__device__ void SubBytes(u_char **state_t,u_char* d_sbox){
	for(int i = 0 ; i < 4; i++)
		for(int j = 0 ; j < 4 ; j++)
			state_t[i][j] = d_sbox[state_t[i][j]];
}

__device__ void ShiftRows(u_char **state_t){
	for(int i = 0 ; i < 4 ; i++)
	{
		for(int j = i ; j > 0  ; j--)
		{
			u_char temp = state_t[i][0];
			for(int j = 1 ;j < 4; j++)
			{
				state_t[i][j-1] = state_t[i][j];
			}

			state_t[i][3] = temp;
		}
	}
}

void KeyExpansionFunc(Key *Expanded_Key_arr, Key k_init){
	Word temp;
	int round_no = 1;

	for(int i = 0 ; i < 4 ; i++)
	{
		Expanded_Key_arr[0].word[i] = k_init.word[i];
	}

	for(int m = 1;  m < 11 ; m++)
	{
		for(int i = 0; i < 4 ; i++)
		{
			if(i != 0)
				temp = Expanded_Key_arr[m].word[i-1];

			if(i == 0)
			{

				temp = Expanded_Key_arr[m-1].word[3];

				//Left Circular Shift (ByteWise)
				u_char temp_byte = temp.byte[0];
				for(int j = 1 ; j < 4 ; j++)
				{
					temp.byte[j-1] = temp.byte[j];
				}
				temp.byte[3] = temp_byte;
				//Getting S box Value
				for(int j = 0 ;  j < 4; j++)
				{
					temp.byte[j] = s_box[temp.byte[j]];
				}
				temp.byte[0] = temp.byte[0]^Rcon[round_no];
				round_no++;
			}

			for(int j = 0 ;  j < 4; j++)
			{
					Expanded_Key_arr[m].word[i].byte[j] = (Expanded_Key_arr[m-1].word[i].byte[j])^temp.byte[j];
			}
		}
	}
}

void PrintKey(int i, Key *Expanded_Key_arr){

		for(int j =0 ; j < 4; j++)
		{
			for(int k =0 ; k < 4; k++)
			{
				printf(" %2X ",Expanded_Key_arr[i].word[j].byte[k]);
			}
			printf("\n");
		}
		printf("\n\n\n");
}

__device__ void MixColumns(u_char **state_t,u_char* d_mixMat){
	u_char **temp_state = (u_char **)malloc(sizeof(u_char *)*4);
	for(int i = 0; i < 4; i++)
		 temp_state[i] = (u_char *)malloc(sizeof(u_char)*4);

	for(int i = 0 ; i < 4 ; i++)
	{
		for(int j = 0 ; j < 4; j++)
		{
			int sum = 0 ;
			for(int k = 0 ; k < 4 ; k++)
			{
				sum = sum^gmul(d_mixMat[i*4+k],state_t[k][j]);
			}
			temp_state[i][j] = sum;
		}
	}
	for(int i = 0 ; i < 4 ; i++)
	{
		for(int j = 0 ; j < 4; j++)
		{
			state_t[i][j] = temp_state[i][j];
		}
	}
}


Key Copy_key_str(char* str)
{
	Key temp_arr;

	int i = 0;

		for(int j = 0 ;  j < 4; j++)
		{
			for(int k = 0 ;  k < 4; k++)
			{
				temp_arr.word[j].byte[k] = str[i];
				i++;
			}
		}

		return temp_arr;
}

__device__ void AddRoundKey(u_char* mod_arr, u_char **state_t, int round){

	for(int i = 0 ; i < 4 ; i++)
	{
		for(int j = 0 ; j < 4; j++)
		{
			state_t[i][j] = state_t[i][j]^mod_arr[round*16+j*4+i];

		}
	}
}

//blockSize,nonceArr,mod_arr, 
__global__ void func(u_char *nonceArr, u_char *mod_arr, u_char *sbox, u_char *mixmat, u_char* input_arr,u_char *ciphertext,int blockSize) {
	int id  = threadIdx.x;

	u_char **state_t = (u_char **)malloc(sizeof(u_char *)*4);
	for(int i = 0; i < 4; i++)
 		state_t[i] = (u_char *) malloc(sizeof(u_char)*4);

 	/*
		//Copy__state_str(blockSize);//Copy To The Temp for Multiplication
		for(int j = 0 ; j < blockSize ; j++){
			state_t[j%4][j/4] = nonceArr[blockSize*id+j];
			//nonceArr[id*blockSize+ j] = state_t[j%4][j/4] ^ input_arr[id*blockSize + j];
		}
		AddRoundKey(Expanded_Key_arr,state_t,0);
		for(int i = 1; i <= 10 ; i++)
		{
			SubBytes(state_t);
			ShiftRows(state_t);

			if(i != 10)
			{
			 	MixColumns(state_t);
			}
			AddRoundKey(Expanded_Key_arr,state_t,i);
		}
		//Copy_Plain(blockSize);

		for(int j = 0 ; j < blockSize ; j++){
			nonceArr[id*blockSize+ j] = state_t[j%4][j/4] ^ input_arr[id*blockSize + j];
		}
	*/

	for(int j = 0 ; j < blockSize ; j++){
    	state_t[j%4][j/4] = nonceArr[blockSize*id+j];
	}
	AddRoundKey(mod_arr,state_t,0);
	for(int i = 1; i <= 10 ; i++)
	{
		SubBytes(state_t,sbox);
		ShiftRows(state_t);

		if(i != 10)
		{
			MixColumns(state_t,mixmat);
		}
		AddRoundKey(mod_arr,state_t,i);
	}

	for(int j = 0; j < blockSize; j++){
		ciphertext[blockSize*id + j] =  state_t[j%4][j/4] ^ input_arr[id*blockSize + j];;
	}

	//printf("%d %2X\n", id, sbox[id]);
}


int main() {

	u_char *d_sbox,*d_nonceArr,*d_mod_arr,*d_mixMat, *d_ciphertext,*d_input_arr;
	int blockSize = 16, numBlocks = 1500;
	u_char Mix_Mat[16] = {0x02,0x03,0x01,0x01,0x01,0x02,0x03,0x01,0x01,0x01,0x02,0x03,0x03,0x01,0x01,0x02};
	char* key_str =(char*) calloc(blockSize+1,sizeof(char));//Key
	u_char *nonceArr = (u_char*)malloc(sizeof(u_char)*(blockSize*numBlocks));
	char* nonce = (char*)calloc(blockSize+1,sizeof(char));
	u_char *ciphertext = (u_char*)malloc(sizeof(u_char)*(blockSize*numBlocks));
    u_char* input_arr =(u_char *) calloc(blockSize*numBlocks,sizeof(u_char));

	Key Expanded_Key_arr[11];


	FILE* f1 = fopen("plain_text.txt","r");
	fread(input_arr,sizeof(u_char),numBlocks*blockSize,f1);

	strcpy(nonce,"Two One Nine Two");

	for(int i = 0 ; i < blockSize; i++)
	{
		nonceArr[i] = nonce[i];
	}
	Generate_Nonce_Text(nonceArr,blockSize,numBlocks);
	strcpy(key_str,"Thats my Kung Fu");//Hardcoding Key
	Key temp_keys = Copy_key_str(key_str);
	KeyExpansionFunc(Expanded_Key_arr, temp_keys);
	u_char* mod_arr  = (u_char*)calloc(11*4*4,sizeof(u_char));
	for(int r = 0 ; r < 11 ; r++)
	{
		for(int i = 0 ; i < 4 ; i++)
		{
		 	for(int j = 0 ; j < 4; j++)
		 	{
		 		mod_arr[r*16+j*4+i] = Expanded_Key_arr[r].word[j].byte[i];

		 	}
		}
	}
	for(int i = 0; i < 11; i++) {
		PrintKey(i,Expanded_Key_arr);
	} 

	//CORRECT TILL HERE

	hipMalloc((void **)&d_input_arr, (blockSize*numBlocks)*sizeof(u_char));
	hipMemcpy(d_input_arr, input_arr, (blockSize*numBlocks) * sizeof(u_char), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_nonceArr, (blockSize*numBlocks)*sizeof(u_char));
	hipMemcpy(d_nonceArr, nonceArr, (blockSize*numBlocks) * sizeof(u_char), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_ciphertext, (blockSize*numBlocks)*sizeof(u_char));
	hipMemcpy(d_ciphertext, ciphertext, (blockSize*numBlocks) * sizeof(u_char), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_mod_arr, 11*4*4*sizeof(u_char));
	hipMemcpy(d_mod_arr, mod_arr, 11*4*4*sizeof(u_char), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_sbox, 256*sizeof(u_char));
	hipMemcpy(d_sbox, s_box, 256 * sizeof(u_char), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_mixMat, 16*sizeof(u_char));
	hipMemcpy(d_mixMat, Mix_Mat, 16 * sizeof(u_char), hipMemcpyHostToDevice);
	func<<<1, 11>>>(d_nonceArr,d_mod_arr,d_sbox,d_mixMat,d_input_arr,d_ciphertext,blockSize);
	hipMemcpy(ciphertext, d_ciphertext, (blockSize*numBlocks) * sizeof(u_char), hipMemcpyDeviceToHost);
	for(int i = 0 ; i < blockSize*numBlocks ; i++)
		{
			if(i%16 == 0)
				printf("\n");

			printf(" %.2X ",ciphertext[i]);
		}
	hipFree(d_sbox);
}